
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

/********************************************************************
CUDA Kernel
*********************************************************************/
__global__ void matrixMul (float* C, float* A, float* B, int TA)
{
    /* calcul des coordonnees du point de C a calculer */
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    /* calcul de C[i][j] */
    int cc = 0;
    for (int k = 0; k < TA; ++ k)
        cc += A[i * TA + k] * B[k * TA + j];

    /* stockage */
    C[i * TA + j] += cc;
}


/********************************************************************
Programme main
*********************************************************************/

int main (int argc, char** argv)
{
    int i, j, TM, GRID_SIZE_X, GRID_SIZE_Y, BLOCK_SIZE_X, BLOCK_SIZE_Y;
    hipError_t cerror;

    const int TH_BLOCK_X = 128;
    const int TH_BLOCK_Y = 2;

    /* pour le calcul du temps de traitement sur GPU */
    float tc;
    hipEvent_t depart, arret;

    hipEventCreate(&depart);
    hipEventCreate(&arret);

    /* valeurs par defaut */
    TM = 2048;

    /* TM peut etre lu comme arg1 de la commande */
    if (argc > 1) {
        TM = atoi(argv[1]);
    }

    GRID_SIZE_X = TM / TH_BLOCK_X;
    GRID_SIZE_Y = TM / TH_BLOCK_Y;
    BLOCK_SIZE_X = TH_BLOCK_X;
    BLOCK_SIZE_Y = TH_BLOCK_Y;

    /* definiton de la grille et des blocs */
    dim3 grid(GRID_SIZE_X, GRID_SIZE_Y);
    dim3 block(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    printf("taille grille : %d - %d\n", GRID_SIZE_X, GRID_SIZE_Y);
    printf("taille bloc   : %d - %d\n", BLOCK_SIZE_X, BLOCK_SIZE_Y);

    /* allocation des matrices sur CPU */
    unsigned int msize_A = TM * TM * sizeof(float);
    unsigned int msize_B = TM * TM * sizeof(float);
    unsigned int msize_C = TM * TM * sizeof(float);

    float* h_A = (float*) malloc(msize_A);
    float* h_B = (float*) malloc(msize_B);
    float* h_C = (float*) malloc(msize_C);

    /* initialisation des matrices avec des valeurs permettant de verifier le resultat*/
    for (i = 0; i < TM; i++){
        for (j = 0; j < TM; j++){
            h_A[i * TM + j] = 1.0;
            h_B[i * TM + j] = 1.0;
            h_C[i * TM + j] = 0.0;

            if (i == j) {
                h_A[i * TM + j] = (float) (i + 1);
                h_B[i * TM + j] = (float) (i + 1);
            }
        }
    }

    /* allocation des matrices sur GPU */
    float *d_A; hipMalloc((void**) &d_A, msize_A);
    float *d_B; hipMalloc((void**) &d_B, msize_B);
    float *d_C; hipMalloc((void**) &d_C, msize_C);

    /* mesure du temps : top depart */
    hipEventRecord(depart, 0);

    /* copie des matrives A et B depuis le CPU vers le GPU */
    hipMemcpy(d_A, h_A, msize_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, msize_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, msize_C, hipMemcpyHostToDevice);

    /* lancement des threads */
    matrixMul<<< grid, block >>>(d_C, d_A, d_B, TM);

    /* Recuperation valeur de retour GPU */
    cerror = hipGetLastError();
    printf(" retour %d \n", (int) cerror);

    /* copie de la matrice C depuis le GPU */
    hipMemcpy(h_C, d_C, msize_C, hipMemcpyDeviceToHost);

    /* mesure du temps */
    hipEventRecord(arret, 0);
    hipEventSynchronize(arret);
    hipEventElapsedTime(&tc, depart, arret);
    printf("Temps calcul : %f seconde\n", tc / 1000.0);


    /* verification du resultat */
    for (i = 0; i < TM; i++) {
        for (j = 0; j < TM; j++) {
            if ((i == j) && (h_C[i * TM + j] != (float)((i + 1) * (i + 1) + TM - 1))) {
                printf("Erreur i: %d j: %d %f\n", i, j, h_C[i * TM + j] ); exit(1);
            }
            else if ((i != j) && (h_C[i * TM + j] != (float)(i + j + TM))) {
                printf("Erreur i: %d j: %d\n", i, j);
                exit(1);
            }
        }
    }

    /* liberation de la memoire */
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(depart);
    hipEventDestroy(arret);

}
